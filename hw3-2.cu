#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


//======================
#define DEV_NO 0
hipDeviceProp_t prop;

const int INF = ((1 << 30) - 1);
int n, m, n_old;
int *Dist, *dDist;
#define B 32   //block size




void input(char* infile) {
    FILE* file = fopen(infile, "rb");
    fread(&n_old, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);

    n = (n_old + B - 1) / B * B;

    Dist = (int*)malloc(n*n*sizeof(int));

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (i == j && i < n_old) {
                Dist[i*n+j] = 0;
            } else {
                Dist[i*n+j] = INF;
            }
        }
    }

    int pair[3];
    for (int i = 0; i < m; ++i) {
        fread(pair, sizeof(int), 3, file);
        Dist[pair[0]*n + pair[1]] = pair[2];
    }
    fclose(file);

    
}

void output(char* outFileName) {
    FILE* outfile = fopen(outFileName, "w");
    for (int i = 0; i < n_old; ++i) {
        fwrite(&Dist[i*n], sizeof(int), n_old, outfile);
    }
    fclose(outfile);
}

int ceil(int a, int b) { return (a + b - 1) / b; }


__global__ void phase1(int *dDist, int n, int r){
    __shared__ int s[B*B];

    int j = threadIdx.x;
    int i = threadIdx.y;

    // block id = (r,r)  p.s. block refer to partition in dist matrix, not kernel block 
    int actual_start_i = r * B;
    int actual_start_j = r * B;


    s[ i*B + j ] = dDist[ (i+actual_start_i)*n + (j+actual_start_j)];
    
    for (int k = 0; k < B ; ++k){
        __syncthreads();
        s[ i*B + j ] = min( s[ i*B + j ] , s[ i*B + k ]+s[ k*B + j ]);
    }

    dDist[ (i+actual_start_i)*n + (j+actual_start_j)] = s[ i*B + j ];
}

__global__ void phase2(int *dDist, int n, int r, int round, bool row){

    __shared__ int s[B*B];
    __shared__ int block_r_r[B*B];

    int j = threadIdx.x;
    int i = threadIdx.y;

    int actual_start_i;
    int actual_start_j;

    
    if(row == 1){
        actual_start_i = r * B;
        actual_start_j = ((r+1 + blockIdx.x) % round) * B;
    }
    else{
        actual_start_j = r * B;
        actual_start_i = ((r+1 + blockIdx.x) % round) * B;
    }

    s[ i*B + j ] = dDist[ (i+actual_start_i)*n + (j+actual_start_j)];
    block_r_r[ i*B + j] = dDist[ (i+ r*B )*n + (j+ r*B )];  
    int tmp = dDist[ (i+actual_start_i)*n + (j+actual_start_j)];                 
    
    if(row == 1){
        for (int k = 0; k < B ; ++k){
            
            __syncthreads();
            s[ i*B + j ] = tmp;
            tmp = min( s[ i*B + j ] , block_r_r[ i*B + k ]+s[ k*B + j ]);
        }
    }
    else{
        for (int k = 0; k < B ; ++k){
            
            __syncthreads();
            s[ i*B + j ] = tmp;
            tmp = min( s[ i*B + j ] , s[ i*B + k ]+block_r_r[ k*B + j ]);
        }
    }
    

    dDist[ (i+actual_start_i)*n + (j+actual_start_j)] = tmp;
}

__global__ void phase3(int *dDist, int n, int r, int round){

    __shared__ int block_x_r[B*B];
    __shared__ int block_r_x[B*B];

    int j = threadIdx.x;
    int i = threadIdx.y;
  
    int actual_start_i = ((r+1 + blockIdx.y) % round) * B;
    int actual_start_j = ((r+1 + blockIdx.x) % round) * B;

    block_x_r[ i*B + j ] = dDist[ (i+actual_start_i)*n + (j+r*B)];
    block_r_x[ i*B + j] = dDist[ (i+ r*B )*n + (j+ actual_start_j )];                       
    
    int tmp = dDist[ (i+actual_start_i)*n + (j+ actual_start_j)];

    for (int k = 0; k < B ; ++k){
        __syncthreads();
        tmp = min( tmp , block_x_r[ i*B + k ]+block_r_x[ k*B + j ]);
    }
    
    dDist[ (i+actual_start_i)*n + (j+ actual_start_j)] = tmp;
    
}

void block_FW(){
    int round = n/B;
    for (int r = 0; r < round; ++r){
        phase1<<<1,dim3(B,B)>>>(dDist, n, r);
        hipStream_t stream1, stream2;
        hipStreamCreate(&stream1); hipStreamCreate(&stream2);
        phase2<<< round -1 ,dim3(B,B),0,stream1>>>(dDist, n, r, round, 0);
        phase2<<< round -1 ,dim3(B,B),0,stream2>>>(dDist, n, r, round, 1);
        hipDeviceSynchronize();
        phase3<<<dim3(round -1, round -1),dim3(B,B)>>>(dDist, n, r, round);
    }
}



int main(int argc, char* argv[]) {


    input(argv[1]);
    printf("n = %d\n",n_old);
    hipHostRegister(Dist, n*n*sizeof(int), hipHostRegisterDefault);
    
    hipMalloc(&dDist, n*n*sizeof(int));
   
    hipMemcpy(dDist, Dist, n*n*sizeof(int), hipMemcpyHostToDevice);
    

    //cudaGetDeviceProperties(&prop, DEV_NO);
    //printf("maxThreasPerBlock = %d, sharedMemPerBlock = %d", prop.maxThreadsPerBlock, prop.sharedMemPerBlock);
    //maxThreasPerBlock = 1024, sharedMemPerBlock = 49152(12288 int)

    block_FW();
    
    hipMemcpy(Dist, dDist, n*n*sizeof(int), hipMemcpyDeviceToHost);
   
    output(argv[2]);
    return 0;
}

